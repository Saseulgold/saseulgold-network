#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>

// SHA-256 상수 K 테이블
__device__ __constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// 비트 회전 함수
__device__ inline uint32_t rotr(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

// 문자열을 숫자로 변환 (단순히 0-9 숫자만 처리)
__device__ void uint_to_str(uint64_t num, char *str, int max_len) {
    int i = max_len - 1;
    str[i] = '\0';
    i--;
    if (num == 0) {
        str[i] = '0';
        i--;
    }
    while (num > 0 && i >= 0) {
        str[i] = '0' + (num % 10);
        num /= 10;
        i--;
    }
    // Shift the string to the start
    int start = i + 1;
    int j = 0;
    while (str[start] != '\0') {
        str[j++] = str[start++];
    }
    str[j] = '\0';
}

// SHA-256 패딩 처리
__device__ int sha256_pad(const char *input, int input_len, uint8_t *padded, int padded_size) {
    // 현재 구현은 간단히 64바이트 메시지만 처리 (길이가 55바이트 이하일 때)
    if (input_len > 55) return -1;

    // 복사
    for(int i=0; i<input_len; i++) {
        padded[i] = input[i];
    }
    // 패딩 시작
    padded[input_len] = 0x80;
    for(int i=input_len+1; i<56; i++) {
        padded[i] = 0x00;
    }
    // 메시지 길이 (비트 단위)
    uint64_t bit_len = input_len * 8;
    padded[56] = (bit_len >> 56) & 0xFF;
    padded[57] = (bit_len >> 48) & 0xFF;
    padded[58] = (bit_len >> 40) & 0xFF;
    padded[59] = (bit_len >> 32) & 0xFF;
    padded[60] = (bit_len >> 24) & 0xFF;
    padded[61] = (bit_len >> 16) & 0xFF;
    padded[62] = (bit_len >> 8) & 0xFF;
    padded[63] = bit_len & 0xFF;

    return 0;
}

// SHA-256 변환 함수
__device__ void sha256_transform(const uint8_t *data, uint32_t *state) {
    uint32_t w[64];
    // 메시지 파싱 및 스케줄 생성
    for (int i = 0; i < 16; ++i) {
        w[i] = (data[i * 4] << 24) |
               (data[i * 4 + 1] << 16) |
               (data[i * 4 + 2] << 8) |
               (data[i * 4 + 3]);
    }

    for (int i = 16; i < 64; ++i) {
        uint32_t s0 = rotr(w[i - 15], 7) ^ rotr(w[i - 15], 18) ^ (w[i - 15] >> 3);
        uint32_t s1 = rotr(w[i - 2], 17) ^ rotr(w[i - 2], 19) ^ (w[i - 2] >> 10);
        w[i] = w[i - 16] + s0 + w[i - 7] + s1;
    }

    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];
    uint32_t e = state[4], f = state[5], g = state[6], h = state[7];

    for (int i = 0; i < 64; ++i) {
        uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
        uint32_t ch = (e & f) ^ (~e & g);
        uint32_t temp1 = h + S1 + ch + k[i] + w[i];
        uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        uint32_t temp2 = S0 + maj;
        h = g;
        g = f;
        f = e;
        e = d + temp1;
        d = c;
        c = b;
        b = a;
        a = temp1 + temp2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

// 논스 기반 SHA-256 CUDA 커널
__global__ void sha256_nonce_kernel(const char *seed, int seed_len, uint32_t *output, uint64_t num_threads, uint64_t start_nonce, uint64_t *nonce_found) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_threads) return;

    uint64_t nonce = start_nonce + (uint64_t)idx;

    // 현재 nonce_found가 아직 초기값(0xFFFFFFFFFFFFFFFF)인지 확인
    if (atomicCAS((unsigned long long int*)nonce_found, 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFFFFFFFFFF) == 0xFFFFFFFFFFFFFFFF) {
        // 문자열 결합: seed + "," + nonce + ","
        char nonce_str[25];
        uint_to_str(nonce, nonce_str, 25);

        // ',' 삽입
        int comma_pos = seed_len;
        char full_input[64] = {0};
        for(int i=0; i<seed_len; i++) {
            full_input[i] = seed[i];
        }
        full_input[comma_pos++] = ',';
        // nonce_str 뒤에 ',' 추가
        int j = 0;
        while(nonce_str[j] != '\0' && comma_pos < 63) {
            full_input[comma_pos++] = nonce_str[j++];
        }
        full_input[comma_pos++] = ',';
        full_input[comma_pos] = '\0';
        int full_len = comma_pos; // 실제 입력 길이

        // 패딩
        uint8_t padded[64];
        if (sha256_pad(full_input, full_len, padded, 64) != 0) {
            // 패딩 실패 (메시지 길이 초과)
            return;
        }

        // 초기 해시 상태
        uint32_t state[8] = {
            0x6a09e667,
            0xbb67ae85,
            0x3c6ef372,
            0xa54ff53a,
            0x510e527f,
            0x9b05688c,
            0x1f83d9ab,
            0x5be0cd19
        };

        // SHA-256 변환
        sha256_transform(padded, state);
        // 특정 조건 확인 (예: 해시 값 상위 20비트가 0)
        // if (state[0] < 0x00000000 && state[1] < 0x80000000) { // 예시: 상위 20비트가 0이 되도록
        if (state[0] < 0x00000001) { // 예시: 상위 20비트가 0이 되도록
            // 최초로 찾은 nonce만 기록
            if (atomicCAS((unsigned long long int*)nonce_found, 0xFFFFFFFFFFFFFFFF, nonce) == 0xFFFFFFFFFFFFFFFF) {
                // 해시 결과 저장
                for(int i=0; i<8; i++) {
                    output[i] = state[i];
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <epoch>\n", argv[0]);
        return 1;
    }

    const char *epoch = argv[1];
    int epoch_len = strlen(epoch);
    const char seed_prefix[] = "blk-";  // 예시로 seed prefix를 "blk-"로 설정
    int seed_prefix_len = strlen(seed_prefix);

    // seed = seed_prefix + epoch
    char seed[64] = {0};
    for(int i=0; i<seed_prefix_len; i++) {
              seed[i] = seed_prefix[i];
    }
    for(int i=0; i<epoch_len; i++) {
        seed[seed_prefix_len + i] = epoch[i];
    }
    int seed_len = seed_prefix_len + epoch_len;

    const uint64_t num_threads = 1024 * 1024;  // 스레드 수 (예: 1,048,576)
    uint32_t *h_output = (uint32_t *)malloc(8 * sizeof(uint32_t));
    uint64_t h_nonce_found = 0xFFFFFFFFFFFFFFFF;

    char *d_seed;
    uint32_t *d_output;
    uint64_t *d_nonce_found;

    hipMalloc(&d_seed, seed_len * sizeof(char));
    hipMalloc(&d_output, 8 * sizeof(uint32_t));
    hipMalloc(&d_nonce_found, sizeof(uint64_t));

    hipMemcpy(d_seed, seed, seed_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_nonce_found, &h_nonce_found, sizeof(uint64_t), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int num_blocks_cuda = (num_threads + threads_per_block - 1) / threads_per_block;

    uint64_t start_nonce = 0;
    bool found = false;

    while(!found) {
        // 현재 nonce_found를 초기값으로 재설정
        h_nonce_found = 0xFFFFFFFFFFFFFFFF;
        hipMemcpy(d_nonce_found, &h_nonce_found, sizeof(uint64_t), hipMemcpyHostToDevice);

        // 커널 실행
        sha256_nonce_kernel<<<num_blocks_cuda, threads_per_block>>>(d_seed, seed_len, d_output, num_threads, start_nonce, d_nonce_found);
        hipDeviceSynchronize();

        // nonce_found 복사
        hipMemcpy(&h_nonce_found, d_nonce_found, sizeof(uint64_t), hipMemcpyDeviceToHost);

        if (h_nonce_found != 0xFFFFFFFFFFFFFFFF) {
            // nonce가 발견됨
            printf("Final Nonce found: %llu\n", h_nonce_found);
            // 해당 nonce의 해시값을 호스트로 복사하여 출력
            hipMemcpy(h_output, d_output, 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
            printf("Final Hash: %08x %08x %08x %08x %08x %08x %08x %08x\n",
                   h_output[0], h_output[1], h_output[2], h_output[3],
                   h_output[4], h_output[5], h_output[6], h_output[7]);
            found = true;
        } else {
            // nonce가 발견되지 않음, 시작 nonce를 증가시킴
            start_nonce += num_threads;
            printf("Nonce not found in range %llu to %llu. Trying next range...\n", start_nonce, start_nonce + num_threads - 1);
        }
    }

    hipFree(d_seed);
    hipFree(d_output);
    hipFree(d_nonce_found);

    free(h_output);

    return 0;
}

