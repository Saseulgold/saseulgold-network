#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>

// SHA-256 Constant K table
__device__ __constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// Bit rotation function
__device__ inline uint32_t rotr(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

// Converting strings to numbers (simply processing 0-9 numbers only)
__device__ void uint_to_str(uint64_t num, char *str, int max_len) {
    int i = max_len - 1;
    str[i] = '\0';
    i--;
    if (num == 0) {
        str[i] = '0';
        i--;
    }
    while (num > 0 && i >= 0) {
        str[i] = '0' + (num % 10);
        num /= 10;
        i--;
    }
    // Shift the string to the start
    int start = i + 1;
    int j = 0;
    while (str[start] != '\0') {
        str[j++] = str[start++];
    }
    str[j] = '\0';
}

// SHA-256 Padding treatment
__device__ int sha256_pad(const char *input, int input_len, uint8_t *padded, int padded_size) {
    // The current implementation simply handles 64-byte messages (when the length is 55 bytes or less)
    if (input_len > 55) return -1;

    // duplication
    for(int i=0; i<input_len; i++) {
        padded[i] = input[i];
    }
    // Padding, start
    padded[input_len] = 0x80;
    for(int i=input_len+1; i<56; i++) {
        padded[i] = 0x00;
    }
    // Message Length (in bits)
    uint64_t bit_len = input_len * 8;
    padded[56] = (bit_len >> 56) & 0xFF;
    padded[57] = (bit_len >> 48) & 0xFF;
    padded[58] = (bit_len >> 40) & 0xFF;
    padded[59] = (bit_len >> 32) & 0xFF;
    padded[60] = (bit_len >> 24) & 0xFF;
    padded[61] = (bit_len >> 16) & 0xFF;
    padded[62] = (bit_len >> 8) & 0xFF;
    padded[63] = bit_len & 0xFF;

    return 0;
}

// SHA-256 Transformation function
__device__ void sha256_transform(const uint8_t *data, uint32_t *state) {
    uint32_t w[64];
    // Parsing messages and generating
    for (int i = 0; i < 16; ++i) {
        w[i] = (data[i * 4] << 24) |
               (data[i * 4 + 1] << 16) |
               (data[i * 4 + 2] << 8) |
               (data[i * 4 + 3]);
    }

    for (int i = 16; i < 64; ++i) {
        uint32_t s0 = rotr(w[i - 15], 7) ^ rotr(w[i - 15], 18) ^ (w[i - 15] >> 3);
        uint32_t s1 = rotr(w[i - 2], 17) ^ rotr(w[i - 2], 19) ^ (w[i - 2] >> 10);
        w[i] = w[i - 16] + s0 + w[i - 7] + s1;
    }

    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];
    uint32_t e = state[4], f = state[5], g = state[6], h = state[7];

    for (int i = 0; i < 64; ++i) {
        uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
        uint32_t ch = (e & f) ^ (~e & g);
        uint32_t temp1 = h + S1 + ch + k[i] + w[i];
        uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        uint32_t temp2 = S0 + maj;
        h = g;
        g = f;
        f = e;
        e = d + temp1;
        d = c;
        c = b;
        b = a;
        a = temp1 + temp2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

// Nonce-based SHA-256 CUDA Kernel
__global__ void sha256_nonce_kernel(const char *seed, int seed_len, uint32_t *output, uint64_t num_threads, uint64_t start_nonce, uint64_t *nonce_found) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_threads) return;

    uint64_t nonce = start_nonce + (uint64_t)idx;

    // Verify that the current nonce_found is still an initial value (0xFFFFFFFFFFFFFFFFFFFFFFFFFFFF)
    if (atomicCAS((unsigned long long int*)nonce_found, 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFFFFFFFFFF) == 0xFFFFFFFFFFFFFFFF) {
        // String combination: seed + "," + nonce + ","
        char nonce_str[25];
        uint_to_str(nonce, nonce_str, 25);

        // ',' insert
        int comma_pos = seed_len;
        char full_input[64] = {0};
        for(int i=0; i<seed_len; i++) {
            full_input[i] = seed[i];
        }
        full_input[comma_pos++] = ',';
        // Add ',' after nonce_str
        int j = 0;
        while(nonce_str[j] != '\0' && comma_pos < 63) {
            full_input[comma_pos++] = nonce_str[j++];
        }
        full_input[comma_pos++] = ',';
        full_input[comma_pos] = '\0';
        int full_len = comma_pos; // Actual Input Length

        // Padding
        uint8_t padded[64];
        if (sha256_pad(full_input, full_len, padded, 64) != 0) {
            // Padding failed (message length exceeded)
            return;
        }

        // Initial hash status
        uint32_t state[8] = {
            0x6a09e667,
            0xbb67ae85,
            0x3c6ef372,
            0xa54ff53a,
            0x510e527f,
            0x9b05688c,
            0x1f83d9ab,
            0x5be0cd19
        };

        // SHA-256 conversion
        sha256_transform(padded, state);
        // Check specific conditions (e.g. hash value top 20 bits are 0)
        // if (state[0] < 0x00000000 && state[1] < 0x80000000) {
        if (state[0] < 0x00000001) { // // Example: To ensure that the top 20 bits are zero
            // Record only the first nonce found
            if (atomicCAS((unsigned long long int*)nonce_found, 0xFFFFFFFFFFFFFFFF, nonce) == 0xFFFFFFFFFFFFFFFF) {
                // Save hash results
                for(int i=0; i<8; i++) {
                    output[i] = state[i];
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
	if (argc != 3) {
        printf("Usage: %s <arg1> <arg2>\n", argv[0]);
        return 1;
    }

    const char *arg1 = argv[1];
    const char *arg2 = argv[2];
    int arg1_len = strlen(arg1);
    int arg2_len = strlen(arg2);

    const char seed_prefix[] = "blk-";  // Seed prefix
    int seed_prefix_len = strlen(seed_prefix);
    const char separator = ',';         // Separator between arguments

    // Calculate total seed length: prefix + arg1 + separator + arg2
    int seed_len = seed_prefix_len + arg1_len + 1 + arg2_len;
    if (seed_len >= 64) { // Ensure seed does not exceed buffer size
        printf("Seed length exceeds buffer size.\n");
        return 1;
    }

    // Construct the seed string: "blk-{arg1}-{arg2}"
    char seed[64] = {0};
    int pos = 0;

    // Copy prefix
    memcpy(seed + pos, seed_prefix, seed_prefix_len);
    pos += seed_prefix_len;

    // Copy first argument
    memcpy(seed + pos, arg1, arg1_len);
    pos += arg1_len;

    // Insert separator
    seed[pos++] = separator;

    // Copy second argument
    memcpy(seed + pos, arg2, arg2_len);
    pos += arg2_len;

    // Null-terminate the seed string
    seed[pos] = '\0';

    const uint64_t num_threads = 1024 * 1024;  // Number of Threads (ex: 1,048,576)
    uint32_t *h_output = (uint32_t *)malloc(8 * sizeof(uint32_t));
    uint64_t h_nonce_found = 0xFFFFFFFFFFFFFFFF;

    char *d_seed;
    uint32_t *d_output;
    uint64_t *d_nonce_found;

    hipMalloc(&d_seed, seed_len * sizeof(char));
    hipMalloc(&d_output, 8 * sizeof(uint32_t));
    hipMalloc(&d_nonce_found, sizeof(uint64_t));

    hipMemcpy(d_seed, seed, seed_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_nonce_found, &h_nonce_found, sizeof(uint64_t), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int num_blocks_cuda = (num_threads + threads_per_block - 1) / threads_per_block;

    uint64_t start_nonce = 0;
    bool found = false;

    while(!found) {
        // Reset the current nonce_found to its initial value
        h_nonce_found = 0xFFFFFFFFFFFFFFFF;
        hipMemcpy(d_nonce_found, &h_nonce_found, sizeof(uint64_t), hipMemcpyHostToDevice);

        // Run the kernel
        sha256_nonce_kernel<<<num_blocks_cuda, threads_per_block>>>(d_seed, seed_len, d_output, num_threads, start_nonce, d_nonce_found);
        hipDeviceSynchronize();

        // Copy nonce_found
        hipMemcpy(&h_nonce_found, d_nonce_found, sizeof(uint64_t), hipMemcpyDeviceToHost);

        if (h_nonce_found != 0xFFFFFFFFFFFFFFFF) {
            // Nonce found
            printf("%llu\n", h_nonce_found);
            // Copy and output the hash value of that nonce to the host
            hipMemcpy(h_output, d_output, 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
            printf("%08x%08x%08x%08x%08x%08x%08x%08x\n",
                   h_output[0], h_output[1], h_output[2], h_output[3],
                   h_output[4], h_output[5], h_output[6], h_output[7]);
            found = true;
        } else {
            // Nonce not found, increasing start nonce
            start_nonce += num_threads;
            // printf("Nonce not found in range %llu to %llu. Trying next range...\n", start_nonce, start_nonce + num_threads - 1);
        }
    }

    hipFree(d_seed);
    hipFree(d_output);
    hipFree(d_nonce_found);

    free(h_output);

    return 0;
}

